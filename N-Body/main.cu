#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include "nBodyUtils.cuh"

inline hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
	if (result != hipSuccess)
	{
		fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
		assert(result == hipSuccess);
	}
#endif
	return result;
}


// Basic Parallel N-Body Simulation
__global__ void simulationStep(double3* p, double3* v, double* m, double3* f, size_t numBodies, double dt) {
    size_t i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i >= numBodies) return;

    double3 force = make_double3(0.0, 0.0, 0.0);
    for (size_t j = 0; j < numBodies; j++) {
        if (i != j) {
            double3 diff = make_double3(
                p[j].x - p[i].x,
                p[j].y - p[i].y,
                p[j].z - p[i].z
                );
            double distSqr = diff.x * diff.x + diff.y * diff.y + diff.z * diff.z + 1e-10;
            double dist = sqrt(distSqr);
            if (dist > 0) {
                double F = G * m[i] * m[j] / distSqr;
                force.x += F * diff.x / dist;
                force.y += F * diff.y / dist;
                force.z += F * diff.z / dist;
            }
        }
    }
    f[i] = force;

    v[i].x += dt * f[i].x / m[i];
    v[i].y += dt * f[i].y / m[i];
    v[i].z += dt * f[i].z / m[i];
    p[i].x += dt * v[i].x;
    p[i].y += dt * v[i].y;
    p[i].z += dt * v[i].z;
}

// Sequential CPU N-Body Simulation
void simulationStepCPU(double3* p, double3* v, double* m, double3* f, size_t numBodies, double dt) {
    for (size_t i = 0; i < numBodies; i++) {
        double3 force = make_double3(0.0, 0.0, 0.0);
        for (size_t j = 0; j < numBodies; j++) {
            if (i != j) {
                double3 diff = make_double3(
                    p[j].x - p[i].x,
                    p[j].y - p[i].y,
                    p[j].z - p[i].z
                );
                double distSqr = diff.x * diff.x + diff.y * diff.y + diff.z * diff.z + 1e-10;
                double dist = sqrt(distSqr);
                if (dist > 0) {
                    double F = G * m[i] * m[j] / distSqr;
                    force.x += F * diff.x / dist;
                    force.y += F * diff.y / dist;
                    force.z += F * diff.z / dist;
                }
            }
        }
        f[i] = force;

        v[i].x += dt * f[i].x / m[i];
        v[i].y += dt * f[i].y / m[i];
        v[i].z += dt * f[i].z / m[i];
        p[i].x += dt * v[i].x;
        p[i].y += dt * v[i].y;
        p[i].z += dt * v[i].z;
    }
}

int main() {
    const char* filename = "solar_system.txt";
    const double dt = 0.1;
    const size_t numSteps = 1000000;
    
    size_t numBodies = getNumBodies(filename);
    double3* h_p = new double3[numBodies];
    double3* h_v = new double3[numBodies];
    double* h_m = new double[numBodies];
    loadBodiesFromFile(filename, h_p, h_v, h_m);
    printSimlulationSummary(numBodies, dt, numSteps, G, 256, 256);

    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Move data to GPU
    double3* d_p, *d_v, *d_f;
    double* d_m;

    checkCuda(hipMalloc(&d_p, numBodies * sizeof(double3)));
    checkCuda(hipMalloc(&d_v, numBodies * sizeof(double3)));
    checkCuda(hipMalloc(&d_m, numBodies * sizeof(double)));
    checkCuda(hipMalloc(&d_f, numBodies * sizeof(double3)));

    checkCuda(hipMemcpy(d_p, h_p, numBodies * sizeof(double3), hipMemcpyHostToDevice));
    checkCuda(hipMemcpy(d_v, h_v, numBodies * sizeof(double3), hipMemcpyHostToDevice));
    checkCuda(hipMemcpy(d_m, h_m, numBodies * sizeof(double), hipMemcpyHostToDevice));

    // Record the start event
    hipEventRecord(start);

    // Run CUDA kernel
    int blockSize = 256;
    int numBlocks = std::ceil(numBodies / (float)blockSize);
    size_t step = 0;
    while(step < numSteps) {
        simulationStep<<<numBlocks, blockSize>>>(d_p, d_v, d_m, d_f, numBodies, dt);
        step++;
    }

    // Record the stop event
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // Calculate the elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "\n -> GPU simulation time: " << milliseconds << " ms\n" << std::endl;

    // Retrieve data from GPU
    checkCuda(hipMemcpy(h_p, d_p, numBodies * sizeof(double3), hipMemcpyDeviceToHost));
    checkCuda(hipMemcpy(h_v, d_v, numBodies * sizeof(double3), hipMemcpyDeviceToHost));

    // CPU Simulation
    double3* h_pCPU = new double3[numBodies];
    double3* h_vCPU = new double3[numBodies];
    double* h_mCPU = new double[numBodies];
    double3* h_fCPU = new double3[numBodies];
    loadBodiesFromFile(filename, h_pCPU, h_vCPU, h_mCPU);

    step = 0;
    while(step < numSteps) {
        simulationStepCPU(h_pCPU, h_vCPU, h_mCPU, h_fCPU, numBodies, dt);
        step++;
    }

    // Free memory
    hipFree(d_p);
    hipFree(d_v);
    hipFree(d_m);
    hipFree(d_f);

    // Destroy CUDA events
    hipEventDestroy(start);
    hipEventDestroy(stop);


    // Compare results
    for (int i = 0; i < numBodies; i++) {
        double pos_error = sqrt((h_p[i].x - h_pCPU[i].x) * (h_p[i].x - h_pCPU[i].x) +
                                (h_p[i].y - h_pCPU[i].y) * (h_p[i].y - h_pCPU[i].y) +
                                (h_p[i].z - h_pCPU[i].z) * (h_p[i].z - h_pCPU[i].z));
        double vel_error = sqrt((h_v[i].x - h_vCPU[i].x) * (h_v[i].x - h_vCPU[i].x) +
                                (h_v[i].y - h_vCPU[i].y) * (h_v[i].y - h_vCPU[i].y) +
                                (h_v[i].z - h_vCPU[i].z) * (h_v[i].z - h_vCPU[i].z));
        std::cout << "Body " << i << " Position error: " << pos_error << " Velocity error: " << vel_error << std::endl;
    }


    // Free CPU memory
    delete[] h_p;
    delete[] h_v;
    delete[] h_m;
    delete[] h_pCPU;
    delete[] h_vCPU;
    delete[] h_mCPU;
    delete[] h_fCPU;

    return 0;
}
